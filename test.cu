#include "hip/hip_runtime.h"
 __global__ void scan(
     float *g_odata,
     float *g_idata,
     int n
) {  
    extern __shared__ float temp[];
    
    int id = threadIdx.x;
    int pout = 0, pin = 1;

    temp[pout*n + id] = (id > 0) ? g_idata[id-1] : 0;
    
    __syncthreads();
    
    for (int offset = 1; offset < n; offset *= 2)
    {
        pout = 1 - pout;
        // swap double buffer indices
        pin = 1 - pout;

        if (id >= offset)
        {
            temp[pout*n + id] = temp[pin*n + id - offset] + temp[pout*n+id];
        }
        else 
        {
            temp[pout*n+id] = temp[pin*n+id];
            __syncthreads();
        }
        g_odata[id] = temp[pout*n+id];
        // write output 
    } 
}



__global__ void prescan(
    float *g_odata,
    float *g_idata,
    int n
) {
    extern __shared__ float temp[];

    int thid = threadIdx.x;
    int offset = 1; 
    
    temp[2*thid] = g_idata[2*thid];
    
    for (int d = n>>1; d > 0; d >>= 1)
    // build sum in place up the tree 
    { 
        __syncthreads();
        if (thid < d)
        { 
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1; 
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0)
    {
        temp[n - 1] = 0;
    }

    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d) 
        { 
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1; 
            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }  
    __syncthreads(); 
    g_odata[2*thid] = temp[2*thid];
    // write results to device memory
    g_odata[2*thid+1] = temp[2*thid+1]; 
}

__kernel void prescan(
    __global int *g_odata,
    __global int *g_idata,
    __local int *temp,
    int n
) {
    int thid = get_local_id(0);
    int bid = get_group_id(0);
    int thread_num = get_local_size(0);

    int offset = 1;

    // Make the "empty" spots zeros, so it won't affect the final result.
    if((bid * thread_num + thid) < n)
    {
        temp[thid]  = g_idata[bid * thread_num + thid];
    }else
    {
        temp[thid]  = 0;
    } 

    // build sum in place up the tree
    for (int d = thread_num>>1; d > 0; d >>= 1)
    {
        barrier(CLK_LOCAL_MEM_FENCE);
        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    // clear the last element
    if (thid == 0)
    {
        temp[thread_num - 1] = 0;
    }

    // traverse down tree & build scan
    for (int d = 1; d < thread_num; d *= 2)
    {
        offset >>= 1;
        barrier(CLK_LOCAL_MEM_FENCE);
        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            float t = temp[ai];
            temp[ai]  = temp[ bi];
            temp[bi] += t;
        }
    }
    barrier(CLK_LOCAL_MEM_FENCE);

    g_odata[bid * thread_num + thid] = temp[thid];
}
